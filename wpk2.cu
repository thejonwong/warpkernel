#include "hip/hip_runtime.h"
// General
#include <iostream>
#include <algorithm>
#include <sstream>
#include <assert.h>
// Warpkernel
#include "warpkernel.hpp"

// cusp
#include <cusp/coo_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/detail/timer.h>
#include <cusp/hyb_matrix.h>

// boost
// stats
#include <boost/accumulators/accumulators.hpp>
#include <boost/accumulators/statistics/stats.hpp>
#include <boost/accumulators/statistics/mean.hpp>
#include <boost/accumulators/statistics/min.hpp>

#define ValueType double
#define IndexType int
#define DeviceSpace cusp::device_memory
#define CPUSpace cusp::host_memory

struct rand_float {
  ValueType operator() ()
  {
    return ((ValueType)(rand() % 100))/100. - 0.3;
  }
};


int main(int argc, char *argv[]) {

  std::string matrixfilename = argv[1];
  int ntests = 1;
  if (argc == 3) ntests = atoi(argv[2]);

  int minthreshold = 1;
  int maxthreshold = 1;
  if (argc >= 4) minthreshold = atoi(argv[3]);
  if (argc == 5) maxthreshold = atoi(argv[4]);

  cusp::coo_matrix<IndexType, ValueType, CPUSpace> B;
  cusp::io::read_matrix_market_file(B, matrixfilename.c_str());

  cusp::csr_matrix<IndexType, ValueType, CPUSpace> A = B;

  uint N = A.num_cols;
  uint nz = A.num_entries;

  // open up data file
  std::string filename;
  size_t pos = matrixfilename.find_last_of("/");
  std::string matrixname;
  if (pos != std::string::npos )
    matrixname.assign(matrixfilename.begin()+pos+1, matrixfilename.end());
  else
    matrixname = matrixfilename;
 
  std::string datapath = "./data/" + matrixname + "_results_wpk2.txt";
  if (argc==5)
    datapath = "./data/" + matrixname + "_results_wpk2_" + argv[3] + "_" + argv[4] + ".txt";
  std::cout << "Starting data file = " << datapath << std::endl;
  std::ofstream datafile(datapath.c_str());
  warpkernel::startDatafile(datafile, nz,N,ntests);

  cusp::array1d<ValueType, CPUSpace> x(N,0);
  thrust::generate(x.begin(),x.end(), rand_float());


  cusp::array1d<ValueType, CPUSpace> y(N);

  // setup multiple run mean accumulators
  // find global minimum and maximum

  boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::min>  > wk2all;
  boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::min>  > wk2;
  boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::min>  > wk2no;
  boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::min>  > wk2rex;
  int wk2allblock, wk2block, wk2noblock, wk2rexblock;
  int wk2allth, wk2th, wk2noth, wk2rexth;

  bool lastiter = true;
    // cusp multiplication
    {



      boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::mean>  > statstime;
      cusp::csr_matrix<IndexType, ValueType, DeviceSpace> A1 = A;
      cusp::array1d<ValueType, DeviceSpace> dx = x;
      cusp::array1d<ValueType, DeviceSpace> dy = y;

      cusp::detail::timer cusptimer;
      cusptimer.start();
      for (int t = 0; t < ntests; t++) {
	cusp::multiply(A1,dx,dy);
      }		
      ValueType measuredtime = cusptimer.seconds_elapsed()/ntests;
      statstime(measuredtime);		
      y = dy;
	
      if (lastiter) {
	std::cout << "cusp gpu time " 
		  << std::scientific << boost::accumulators::mean(statstime) << std::endl;
	warpkernel::addData(datafile, "cusp-csr", boost::accumulators::mean(statstime), -1, -1, -1, -1);
      }
    }

    // cusp-hyb
    {
      boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::mean>  > statstime;

      cusp::hyb_matrix<IndexType, ValueType, DeviceSpace> A1 = A;
      cusp::array1d<ValueType, DeviceSpace> dx = x;
      cusp::array1d<ValueType, DeviceSpace> dy = y;

      for (int t = 0; t < ntests; t++) {
	cusp::detail::timer cusptimer;
	cusptimer.start();
	cusp::multiply(A1,dx,dy);
	ValueType measuredtime = cusptimer.seconds_elapsed();
	statstime(measuredtime);
      }

      y = dy;

      if (lastiter) {
	std::cout << "cusp-hyb gpu time " 
		  << std::scientific << boost::accumulators::mean(statstime) << std::endl;
	warpkernel::addData(datafile, "cusp-hyb", boost::accumulators::mean(statstime), -1, -1, -1, -1);
      }
    }


    // test warpkernel2
    {
      cusp::array1d<ValueType, DeviceSpace> dx = x;

      warpkernel::structure2 kernel2;
      kernel2.scan(nz, N, A, (int)N); // use maximum threshold

      int max_nz = kernel2.max_nz;
      int min_nz = kernel2.min_nz;  

      std::cout << "warp kernel 2" << std::endl;

      if (argc > 3) {
	max_nz = maxthreshold;
	min_nz = minthreshold;
      }

      for (int threshold = min_nz; threshold <= max_nz; threshold ++) {
	std::cout << std::endl;
	kernel2.scan(nz, N, A, threshold); // use maximum threshold

	cusp::array1d<IndexType, CPUSpace> reorder_cols = A.column_indices;
	kernel2.reorder_columns_coalesced(reorder_cols);

	cusp::array1d<ValueType, DeviceSpace> dreordered_x;
	kernel2.reorder_x(x, dreordered_x);


	warpkernel::engine<ValueType, IndexType, warpkernel::structure2> eng(kernel2,
									     &(A.values[0]),
									     &(A.column_indices[0]));

	for(int warps_per_block = 2; warps_per_block <= 8; warps_per_block ++) {
	  uint nblocks = (kernel2.nwarps + warps_per_block -1)/warps_per_block;
	  uint blocksize = warps_per_block * WARP_SIZE;    

	  if (nblocks > 65536) continue; // skip if nblocks too high

	  std::cout << std::endl;

	  cusp::array1d<IndexType, DeviceSpace> restore_col = eng.device_colinds;

	  // normal kernel
	  {
	    cusp::array1d<ValueType, DeviceSpace> dy(N,0.);  
	  boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::mean>  > statstime;
	  ValueType totaltime = 0;
	  for (int t = 0; t < ntests; t++) {
	    ValueType measuretime = eng.run<true>(nblocks, blocksize,
						  thrust::raw_pointer_cast(&dx[0]),
						  thrust::raw_pointer_cast(&dy[0]));
	    hipUnbindTexture(x_tex);
	    totaltime += measuretime;
	  }
	  statstime(totaltime/ntests);
	    cusp::array1d<ValueType, CPUSpace> ycheck = dy;
	    if (eng.verify(y,ycheck)) {
	      std::cout << "warpkernel2 (" << nblocks << "," << blocksize << "," << threshold << ") time = " 
			<< std::scientific << boost::accumulators::mean(statstime) << std::endl;
	      std::stringstream kernelname;
	      kernelname << "warpkernel2_" << threshold;
	      warpkernel::addData(datafile, (char *) (kernelname.str()).c_str(), 
				  boost::accumulators::mean(statstime), 
				  kernel2, blocksize);

	      wk2all(boost::accumulators::mean(statstime));
	      wk2(boost::accumulators::mean(statstime));
	      if (boost::accumulators::min(wk2all) == boost::accumulators::mean(statstime)) {
		wk2allblock = blocksize;
		wk2allth = threshold;
	      }
	      if (boost::accumulators::min(wk2) == boost::accumulators::mean(statstime)) {
		wk2block = blocksize;
		wk2th = threshold;
	      }
	    } 
	  }

	  // reordered kernel with rowmap
	  {
	    cusp::array1d<ValueType, DeviceSpace> dy(N,0);  
	    eng.device_colinds = reorder_cols;
	  boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::mean>  > statstime;
	  ValueType totaltime = 0;
	  for (int t = 0; t < ntests; t++) {
	    ValueType measuretime = eng.run<true>(nblocks, blocksize,
						  thrust::raw_pointer_cast(&dreordered_x[0]),
						  thrust::raw_pointer_cast(&dy[0]));
	    hipUnbindTexture(x_tex);
	    totaltime += measuretime;
	  }
	  statstime(totaltime/ntests);
	    cusp::array1d<ValueType, CPUSpace> ycheck = dy;
	    if (eng.verify(y,ycheck) && lastiter) {
	      std::cout << "warpkernel2 reorder (" << nblocks << "," << blocksize << "," << threshold << ") time = " 
			<< std::scientific << boost::accumulators::mean(statstime) << std::endl;

	      std::stringstream kernelname;
	      kernelname << "warpkernel2_" << threshold << "rex";
	      warpkernel::addData(datafile, (char *) (kernelname.str()).c_str(), 
				  boost::accumulators::mean(statstime), 
				  kernel2, blocksize);

	      wk2all(boost::accumulators::mean(statstime));
	    wk2rex(boost::accumulators::mean(statstime));
	    if (boost::accumulators::min(wk2all) == boost::accumulators::mean(statstime)) {
	      wk2allblock = blocksize;
	      wk2allth = threshold;
	    }
	    if (boost::accumulators::min(wk2rex) == boost::accumulators::mean(statstime)) {
	      wk2rexblock = blocksize;
	      wk2rexth = threshold;
	    }

	    } //else exit(1);
	    eng.device_colinds = restore_col;

	  }    

	  // normal kernel counter balances effect of cache
	  {
	    cusp::array1d<ValueType, DeviceSpace> dy(N,0);  
	    ValueType measuretime = eng.run<true>(nblocks, blocksize,
						  thrust::raw_pointer_cast(&dx[0]),
						  thrust::raw_pointer_cast(&dy[0]));
	    hipUnbindTexture(x_tex);
	    cusp::array1d<ValueType, CPUSpace> ycheck = dy;
	  }


	  // reordered kernel without rowmap
	  {
	    cusp::array1d<ValueType, DeviceSpace> dy(N,0);  
	    eng.device_colinds = reorder_cols;
	    boost::accumulators::accumulator_set<ValueType, boost::accumulators::stats<boost::accumulators::tag::mean>  > statstime;
	    ValueType totaltime = 0;
	  for (int t = 0; t < ntests; t++) {
	    ValueType measuretime = eng.run_x<true>(nblocks, blocksize,
						    thrust::raw_pointer_cast(&dreordered_x[0]),
						    thrust::raw_pointer_cast(&dy[0]));
	    hipUnbindTexture(x_tex);
	    totaltime += measuretime;
	  }
	  statstime(totaltime/ntests);
	    cusp::array1d<ValueType, CPUSpace> ycheck = dy;
	    if (eng.verify_x(y,ycheck) && lastiter) {
	      std::cout << "warpkernel2 reorder w/o rowmap (" << nblocks << "," << blocksize << "," << threshold << ") time = " 
			<< std::scientific << boost::accumulators::mean(statstime) << std::endl;

	      std::stringstream kernelname;
	      kernelname << "warpkernel2_" << threshold << "no";
	      warpkernel::addData(datafile, (char *) (kernelname.str()).c_str(), 
				  boost::accumulators::mean(statstime), 
				  kernel2, blocksize);

	      wk2all(boost::accumulators::mean(statstime));
	    wk2no(boost::accumulators::mean(statstime));
	    if (boost::accumulators::min(wk2all) == boost::accumulators::mean(statstime)) {
	      wk2allblock = blocksize;
	      wk2allth = threshold;
	    }
	    if (boost::accumulators::min(wk2no) == boost::accumulators::mean(statstime)) {
	      wk2noblock = blocksize;
	      wk2noth = threshold;
	    }
	    }// else exit(1);
	    eng.device_colinds = restore_col;

	  }

	}
      }
    

    }


    std::stringstream wpk2allname;
    wpk2allname << "wpk2all_" << wk2allth;
    warpkernel::addData(datafile,  (char *) (wpk2allname.str()).c_str(), boost::accumulators::min(wk2all), -1, -1, -1, wk2allblock);
    std::stringstream wpk2name;
    wpk2name << "wpk2_" << wk2th;
    warpkernel::addData(datafile,  (char *)(wpk2name.str()).c_str(), boost::accumulators::min(wk2), -1, -1, -1, wk2block);
    std::stringstream wpk2noname;
    wpk2noname << "wpk2no_" << wk2noth;
    warpkernel::addData(datafile,  (char *)(wpk2noname.str()).c_str(), boost::accumulators::min(wk2no), -1, -1, -1, wk2noblock);
    std::stringstream wpk2rexname;
    wpk2rexname << "wpk2rex_" << wk2rexth;
    warpkernel::addData(datafile,  (char *)(wpk2rexname.str()).c_str(), boost::accumulators::min(wk2rex), -1, -1, -1, wk2rexblock);


}
